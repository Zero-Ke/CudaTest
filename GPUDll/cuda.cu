#include "hip/hip_runtime.h"
#include <>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "GpuDll.h"
#include "CppTimer.h"
#include "iostream"

#define redTo dst[xy].z
#define greenTo dst[xy].y
#define blueTo dst[xy].x
#define grayTo dst[xy].w
int  TestGPU()
{
	int n;
	hipGetDeviceCount(&n);
	std::cout << "Cuda Device Count: " << n << std::endl;
	if (n<0)
	{
		return n;
	}
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int memSize = 10 * sizeof(uchar4);
	uchar4* src_device;
	//Initinaze...
	hipMalloc((void**)&src_device, memSize);
	int a = prop.canMapHostMemory;
	hipFree(src_device);
	std::cout << "Device name: " << a << std::endl;
	return n;
}
__global__ void Kernel_Ratezation(uchar4* pfr, uchar4* pto, int width, int height)
{
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	int grid_width = gridDim.x*blockDim.x;
	int xy = index_y*grid_width + index_x;
	if (index_x>width || index_y>height)
	{
		return;
	}
	int r = pfr[xy].z;
	int g = pfr[xy].y;
	int b = pfr[xy].x;
	int total = r + g + b;
	if (total == 0) total = 1;
	pto[xy].w = (byte)(total / 3);
	pto[xy].z = (byte)(r * 100 / total);
	pto[xy].y = (byte)(g * 100 / total);
	pto[xy].x = (byte)(b * 100 / total);
	
}
__global__ void kernel_Binaryzation(CppColorRange range, uchar4* pfr, uchar4* pto, int width, int height)
{
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	int grid_width = gridDim.x*blockDim.x;
	int xy = index_y*grid_width + index_x;
	if (index_x>width || index_y>height)
	{
		return;
	}
	byte redUp = range.redUp;
	byte redDown = range.redDown;
	byte greenUp = range.greenUp;
	byte greenDown = range.greenDown;
	byte blueUp = range.blueUp;
	byte blueDown = range.blueDown;
	byte grayUp = range.grayUp;
	byte grayDown = range.grayDown;
	int red = pfr[xy].z;
	int green = pfr[xy].y;
	int blue = pfr[xy].x;
	int gray = pfr[xy].w;
	bool choosed = true;
	if (range.redUsed != 0)	{ if (red> redUp) choosed = false; if (red< redDown) choosed = false; }
	if (range.greenUsed != 0)	{ if (green> greenUp) choosed = false;  if (green< greenDown) choosed = false; }
	if (range.greenUsed != 0)	{ if (blue> blueUp) choosed = false; if (blue< blueDown) choosed = false; }
	if (range.greenUsed != 0)	{ if (gray> grayUp) choosed = false; if (gray< grayDown) choosed = false; }
	switch (range.operation)
	{
		case 0:
			if (choosed)	{ pto[xy].w = 0;  }
			break;
		case 1:
			if( choosed )	{ pto[xy].w = 1;}
			break;
		case 2:
			if( !choosed )	{ pto[xy].w= 1;}
			break;
	}
}
__global__ void kernel_BinaryzationShow(CppColorRange* rangeArr, int count, uchar4* dst, int width, int height, int isAll)
{
	int index_x = blockIdx.x*blockDim.x + threadIdx.x;
	int index_y = blockIdx.y*blockDim.y + threadIdx.y;
	int grid_width = gridDim.x*blockDim.x;
	int xy = index_y*grid_width + index_x;
	if (index_x>width || index_y>height)
	{
		return;
	}
	int r = redTo;
	int g = greenTo;
	int b = blueTo;
	int total = r + g + b;
	if (total == 0) total = 1;
	int gray= (byte)(total / 3);
	int red = (byte)(r * 100 / total);
	int green = (byte)(g * 100 / total);
	int blue = (byte)(b * 100 / total);
	//比例化结束,二值化开始
	for (int i = 0; i < count; i++)
	{
		CppColorRange range = *(rangeArr+i);
		bool redUsed = range.redUsed != 0;
		byte redUp = range.redUp;
		byte redDown = range.redDown;
		bool greenUsed = range.greenUsed != 0;
		byte greenUp = range.greenUp;
		byte greenDown = range.greenDown;
		bool blueUsed = range.blueUsed != 0;
		byte blueUp = range.blueUp;
		byte blueDown = range.blueDown;
		bool grayUsed = range.grayUsed != 0;
		byte grayUp = range.grayUp;
		byte grayDown = range.grayDown;
		//printf("%d %d %d %d \n", redUp, greenDown, blueDown, grayUp);
		bool choosed = true;
		if (redUsed)	{ if (red> redUp) choosed = false; if (red< redDown) choosed = false; }
		if (greenUsed)	{ if (green> greenUp) choosed = false;  if (green< greenDown) choosed = false; }
		if (blueUsed)	{ if (blue> blueUp) choosed = false; if (blue< blueDown) choosed = false; }
		if (grayUsed)	{ if (gray> grayUp) choosed = false; if (gray< grayDown) choosed = false; }
		switch (range.operation)
		{
			case 0:	if (choosed)	{ grayTo = 0; } break;
			case 1:	if (choosed)	{ grayTo = 1; }	break;
			case 2:	if (!choosed)	{ grayTo = 1; }	break;
		}
	}
	//二值化结束 ，抽色显示
	if (isAll == 1){ if (grayTo == 0){ redTo = 255; greenTo = 128; blueTo = 128; } }
	else { if (grayTo != 255){ redTo = 0; greenTo = 255; blueTo = 0; } }grayTo = 255;
}
//显示抽色信息
void BinaryShowGPU(CppColorRange*range_host, int count, byte* src_host, byte* dst_host, int width, int height, int isAll)
{
		CppTimer time;
	hipError_t cudastatus;
	
	int memSize = width * height * sizeof(uchar4);
	//uchar4* src_device;
	CppColorRange* range_device;
	//这里面是将原图锁定到内存，并取到锁页内存中的指针Ptr(静态变量，重复利用该指针)，需要的时候方释放该内存
	
	if (!isFirst)
	{
		cudastatus = hipMalloc((void**)&src_device, memSize);
		//方法一：
		//申请host锁页内存----------------耗时1
		cudastatus = hipHostAlloc((void**)&src_host2, memSize, hipHostMallocDefault);
		//拷贝host->host----------------耗时2
		cudastatus = hipMemcpy((void**)src_host2, src_host, memSize, hipMemcpyHostToHost);
		//拷贝host->device
		//方法二
		//host内存注册->锁页内存
		//cudastatus = hipHostRegister(range_host, count*sizeof(CppColorRange), hipHostRegisterMapped);
		//获得设备指针
		//cudastatus = hipHostGetDevicePointer((void **)&range_device, range_host, 0);
		isFirst = true;
	}
	if (src_host2 == NULL)return;
	//这里是将指针Ptr拷贝到GPU上，传输速率是正常的两倍
	cudastatus = hipMemcpy((void**)src_device, src_host2, memSize, hipMemcpyHostToDevice);
	if (cudastatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudastatus));
		return;
	}
	//----------------------------------------------------------------------------/
	cudastatus = hipMalloc((void**)&range_device, count*sizeof(CppColorRange));
	cudastatus = hipMemcpy((void**)range_device, range_host, count*sizeof(CppColorRange), hipMemcpyHostToDevice);
	dim3 threadsPerBlock(32, 32);
	time.Begin();
	dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	kernel_BinaryzationShow << <blocksPerGrid, threadsPerBlock >> >(range_device, count, src_device, width, height, isAll);
	hipDeviceSynchronize();
	//----------------------------------------------------------------------------/
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "kernel_BinaryzationShow launch failed: %s\n", hipGetErrorString(cudaStatus));
		return;
	}
	time.Reset("kernel calculate");
	cudastatus = hipMemcpy((void**)dst_host, src_device, memSize, hipMemcpyDeviceToHost);
	if (cudastatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy launch failed: %s\n", hipGetErrorString(cudastatus));
		return;
	}
	time.Reset("copy to host");
	//hipFree(src_device);
	hipFree(range_device);
}
void Ratezation_GPU(byte* src_host, byte* dst_host, int width, int height)
{
	CppTimer time;
	int length = width * height;
	int memSize = length * sizeof(uchar4);
	uchar4* src_device;
	uchar4* dst_device;
	time.Begin("");
	//Malloc
	hipMalloc((void**)&src_device, memSize);
	hipMalloc((void**)&dst_device, memSize);
	time.Reset("malloc");
	hipError_t cudastatus;
	//Copy To GPU	
	cudastatus=hipMemcpy((void**)src_device, src_host, memSize, hipMemcpyHostToDevice);
	hipMemcpy((void**)dst_device, dst_host, memSize, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(32, 32);
	dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,(height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	time.Reset("copy to device");
	hipDeviceSynchronize();
	Kernel_Ratezation << <blocksPerGrid, threadsPerBlock >> >(src_device, dst_device, width, height);
	time.Reset("kernel calculate");
	hipMemcpy((void**)dst_host, dst_device, memSize, hipMemcpyDeviceToHost);
	time.Reset("copy to host");
	hipFree(src_device);
	hipFree(dst_device);
}

void Binaryzation_GPU(CppColorRange* range_host,byte* src_host, byte* dst_host, int width, int height)
{
	CppTimer time;
	int length = width * height;
	int memSize = length * sizeof(uchar4);
	CppColorRange range_devive = *range_host;
	uchar4* src_device;
	uchar4* dst_device;
	time.Begin();
	//malloc
	hipMalloc((void**)&src_device, memSize);
	hipMalloc((void**)&dst_device, memSize);
	time.Reset("malloc");
	//copy
	hipMemcpy((void**)src_device, src_host, memSize, hipMemcpyHostToDevice);
	hipMemcpy((void**)dst_device, dst_host, memSize, hipMemcpyHostToDevice);
	//handle
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
	time.Reset("Binaryzation:copy to device");
	hipDeviceSynchronize();
	kernel_Binaryzation << <blocksPerGrid, threadsPerBlock >> >(range_devive, src_device, dst_device, width, height);
	time.Reset("Binaryzation:kernel calculate");
	hipMemcpy((void**)dst_host, dst_device, memSize, hipMemcpyDeviceToHost);
	time.Reset("Binaryzation:copy to host");
	hipFree(src_device);
	hipFree(dst_device);
}